
#include <hip/hip_runtime.h>
//
// kernel.cu

///////////////////////// colToGray

__global__ void colToGray_v0(float *inImage, float *outImage, int width, int height, int numChannels) {

    // Determine column and row of thread
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    
    // If within bounds of image, perform grayscale operation.
    if (col < width && row < height) {
      int index = (row * width + col);
      int rgbIndex = index * numChannels;
  
      float r = inImage[rgbIndex];
      float g = inImage[rgbIndex + 1];
      float b = inImage[rgbIndex + 2];
  
      outImage[index] = (0.21 * r + 0.71 * g + 0.07 * b);
    }
}

__global__ void colToGray_v1_0(float *inImage, int imageArea) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= imageArea * 3) return;
    inImage[idx] = inImage[idx] * 0.07;
}

__global__ void colToGray_v1_1(float *inImage, float *outImage, int imageArea) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= imageArea) return;

    int rgbIndex = idx * 3;

    float r = inImage[rgbIndex];
    float g = inImage[rgbIndex + 1];
    float b = inImage[rgbIndex + 2];
  
    outImage[idx] = (3 * r + 10 * g + b);
}

__global__ void colToGray_v2(float *inImage, float *outImage, int imageArea) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= imageArea) return;

    int rgbIndex = idx * 3;

    float r = inImage[rgbIndex];
    float g = inImage[rgbIndex + 1];
    float b = inImage[rgbIndex + 2];
  
    outImage[idx] = (0.21 * r + 0.71 * g + 0.07 * b);
}

__global__ void colToGray_v2_byte(float *inImage, uint8_t *outImage, int imageArea) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= imageArea) return;

    int rgbIndex = idx * 3;

    float r = inImage[rgbIndex];
    float g = inImage[rgbIndex + 1];
    float b = inImage[rgbIndex + 2];
  
    outImage[idx] = (uint8_t)((0.21 * r + 0.71 * g + 0.07 * b) * 255);
}

  // num threads = image area * 3
__global__ void colToGray_v3_0(float *inImage, float *intImage, int imageArea) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= imageArea * 3) return;
    int outIdx = (idx % 3) * imageArea + (idx / 3);
    intImage[outIdx] = inImage[idx];
}

__global__ void colToGray_v3_1(float *intImage, float *outImage, int imageArea) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= imageArea) return;

    float r = intImage[idx];
    float g = intImage[idx + imageArea];
    float b = intImage[idx + imageArea + imageArea];
  
    outImage[idx] = (0.21 * r + 0.71 * g + 0.07 * b);
}

__global__ void colToGray_v4(float *inImage, float *outImage, int imageArea) {
    __shared__ float s[1024];

    int i = threadIdx.x;
    int idx = (threadIdx.x + blockIdx.x * blockDim.x) - blockIdx.x;
    if (idx >= imageArea * 3) return;

    s[i] = inImage[idx];

    __syncthreads();

    if (i < 341) {
        int index = i * 3;
        outImage[i + (blockIdx.x * 341)] = (0.21 * s[index] + 0.71 * s[index+1] + 0.07 * s[index+2]);
    }
}

///////////////////////// cannyEdge

__global__ void cannyEdge_v0_0(float *imageIn, float *gradient, float *angle, int width, int height) {

    // 0 1 2
    // 3   4
    // 5 6 7

    float n[8];

    for (int i = 0; i < 8; i++) {
        n[i] = 0;
    }

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (col < width && row < height) {
        int index = (row * width + col);

        if (col > 0) {
            if (row > 0)
                n[0] = imageIn[index - width - 1];

            if (row < height - 1)
                n[5] = imageIn[index + width - 1]; 

            n[3] = imageIn[index - 1];
        }

        if (col < width - 1) {
            if (row > 0)
                n[2] = imageIn[index - width + 1];

            if (row < height - 1)
                n[7] = imageIn[index + width + 1];

            n[4] = imageIn[index + 1];
        }

        if (row > 0) 
            n[1] = imageIn[index - width];

        if (row < height - 1)
            n[6] = imageIn[index + width];

        float gx = -n[0] + n[2] - (2 * n[3]) + (2 * n[4]) - n[5] + n[7];
        float gy = -n[0] + n[5] - (2 * n[1]) + (2 * n[6]) - n[2] + n[7];

        float grad = sqrtf( pow(gx, 2) + pow(gy, 2) );
        float theta = atan2f( gy, gx );
        theta = theta + (theta < 0) * M_PI;

        gradient[index] = grad;
        angle[index] = theta;
    }
}

__global__ void cannyEdge_v1_0(float *imageIn, float *gradient, float *angle, int width, int height) {

    // 0 1 2
    // 3   4
    // 5 6 7

    __shared__ float n[18][18];

    int col = threadIdx.x + blockIdx.x * blockDim.x; // global col
    int row = threadIdx.y + blockIdx.y * blockDim.y; // global row
    
    if (col >= width || row >= height) return;

    for (int i = 0; i < 18; i++) {
        for (int j = 0; j < 18; j++) {
            n[i][j] = 0;
        }
    }

    __syncthreads();

    int bCol = threadIdx.x; // block col
    int bRow = threadIdx.y; // block row

    int index = (row * width + col);

    n[bCol + 1][bRow + 1] = imageIn[index];

    if (bCol == 0 && col > 0)       n[0][bRow + 1] = imageIn[index - 1]; 
    if (bCol == 15 && col < width - 1)  n[17][bRow + 1] = imageIn[index + 1]; 
    if (bRow == 0 && row > 0)       n[bCol + 1][0] = imageIn[index - width];
    if (bRow == 15 && row < height - 1) n[bCol + 1][17] = imageIn[index + width];

    if (bCol == 0 && bRow == 0 && col > 0 && row > 0)            n[0][0] =  imageIn[index - width - 1];
    if (bCol == 15 && bRow == 0 && col < width - 1 && row > 0)       n[17][0] =  imageIn[index - width + 1];
    if (bCol == 0 && bRow == 15 && col > 0 && row < height - 1)      n[0][17] =  imageIn[index + width - 1];
    if (bCol == 15 && bRow == 15 && col < width - 1 && row < height - 1) n[17][17] =  imageIn[index + width + 1];

    __syncthreads();

    float gx = -n[bCol][bRow] + n[bCol+2][bRow] - (2 * n[bCol][bRow+1]) + (2 * n[bCol+2][bRow+1]) - n[bCol][bRow+2] + n[bCol+2][bRow+2];
    float gy = -n[bCol][bRow] + n[bCol][bRow+2] - (2 * n[bCol+1][bRow]) + (2 * n[bCol+1][bRow+2]) - n[bCol+2][bRow] + n[bCol+2][bRow+2];

    float grad = sqrtf( pow(gx, 2) + pow(gy, 2) );
    float theta = atan2f( gy, gx );
    theta = theta + (theta < 0) * M_PI;

    gradient[index] = grad;
    angle[index] = theta;
}

__global__ void cannyEdge_v2_0(float *imageIn, float2 *out, int width, int height) {

    // 0 1 2
    // 3   4
    // 5 6 7

    float n[8];

    for (int i = 0; i < 8; i++) {
        n[i] = 0;
    }

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (col < width && row < height) {
        int index = (row * width + col);

        if (col > 0) {
            if (row > 0)
                n[0] = imageIn[index - width - 1];

            if (row < height - 1)
                n[5] = imageIn[index + width - 1]; 

            n[3] = imageIn[index - 1];
        }

        if (col < width - 1) {
            if (row > 0)
                n[2] = imageIn[index - width + 1];

            if (row < height - 1)
                n[7] = imageIn[index + width + 1];

            n[4] = imageIn[index + 1];
        }

        if (row > 0) 
            n[1] = imageIn[index - width];

        if (row < height - 1)
            n[6] = imageIn[index + width];

        float gx = -n[0] + n[2] - (2 * n[3]) + (2 * n[4]) - n[5] + n[7];
        float gy = -n[0] + n[5] - (2 * n[1]) + (2 * n[6]) - n[2] + n[7];

        float grad = pow(gx, 2) + pow(gy, 2);
        float theta = atan2f( gy, gx );
        theta = theta + (theta < 0) * M_PI;

        float2 val = make_float2(grad, theta);

        out[index] = val;
    }
}

__global__ void cannyEdge_v2_1(float2 *in, float *edgemap, int width, int height, float lowThresh) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (col < width && row < height) {
        int index = (row * width + col);

        float grad = in[index].x;
        float theta = in[index].y;

        float max = 1;

        if ( (theta < M_PI / 8) || (theta > (7 * M_PI / 8)) ) {
            if (col > 0 && in[index - 1].x > grad)
                max = 0;
            if (col < width - 1 && in[index + 1].x > grad)
                max = 0;
        }
        else if ( theta < (3 * M_PI / 8) ) {
            if (row > 0 && col > 0 && in[index - width - 1].x > grad)
                max = 0;
            if (row < height - 1 && col < width - 1 && in[index + width + 1].x > grad)
                max = 0;
        }
        else if ( theta < (5 * M_PI / 8) ) {
            if (row > 0 && in[index - width].x > grad)
                max = 0;
            if (row < height - 1 && in[index + width].x > grad)
                max = 0;
        }
        else {
            if (row > 0 && col < width - 1 && in[index - width + 1].x > grad)
                max = 0;
            if (row < height - 1 && col > 0 && in[index + width - 1].x > grad)
                max = 0;
        }

        if (grad < pow(lowThresh, 2)) {
            max = 0;
        }

        edgemap[index] = max;
    }
}

__global__ void cannyEdge_v0_1(float *gradient, float *angle, float *edgemap, int width, int height, float lowThresh) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (col < width && row < height) {
        int index = (row * width + col);

        float grad = gradient[index];
        float theta = angle[index];

        float max = 1;

        if ( (theta < M_PI / 8) || (theta > (7 * M_PI / 8)) ) {
            if (col > 0 && gradient[index - 1] > grad)
                max = 0;
            if (col < width - 1 && gradient[index + 1] > grad)
                max = 0;
        }
        else if ( theta < (3 * M_PI / 8) ) {
            if (row > 0 && col > 0 && gradient[index - width - 1] > grad)
                max = 0;
            if (row < height - 1 && col < width - 1 && gradient[index + width + 1] > grad)
                max = 0;
        }
        else if ( theta < (5 * M_PI / 8) ) {
            if (row > 0 && gradient[index - width] > grad)
                max = 0;
            if (row < height - 1 && gradient[index + width] > grad)
                max = 0;
        }
        else {
            if (row > 0 && col < width - 1 && gradient[index - width + 1] > grad)
                max = 0;
            if (row < height - 1 && col > 0 && gradient[index + width - 1] > grad)
                max = 0;
        }

        if (grad < lowThresh) {
            max = 0;
        }

        edgemap[index] = max;
    }
}

__global__ void cannyEdge_v3(float *imageIn, float *edgemap, int width, int height, int lowThresh) {

    // // 0 1 2
    // // 3   4
    // // 5 6 7

    // Part 1: calculate gradient and angle

    __shared__ int ns[64][8];

    int col = (threadIdx.x + blockIdx.x * blockDim.x) - (4 * blockIdx.x) - 2;
    int row = (threadIdx.y + blockIdx.y * blockDim.y) - (4 * blockIdx.y) - 2;
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int index = row * width + col;

    if (col >= 0 && row >= 0 && col < width && row < height)
        ns[tidx][tidy] = (int)(imageIn[index] * 100);
    else
        ns[tidx][tidy] = 0;

    __syncthreads();

    if (tidx >= 1 && tidy >= 1 && tidx <= blockDim.x-2 && tidy <= blockDim.y-2) {

        int gx = -ns[tidx-1][tidy-1] + ns[tidx+1][tidy-1] - (2 * ns[tidx-1][tidy]) + (2 * ns[tidx+1][tidy]) - ns[tidx-1][tidy+1] + ns[tidx+1][tidy+1];
        int gy = -ns[tidx-1][tidy-1] + ns[tidx-1][tidy+1] - (2 * ns[tidx][tidy-1]) + (2 * ns[tidx][tidy+1]) - ns[tidx+1][tidy-1] + ns[tidx+1][tidy+1];

        __syncthreads();

        int grad = (int)(sqrtf( powf(gx, 2) + powf(gy, 2) ));
        int theta = (int)(atan2f( gy, gx ) * 180 / M_PI);
        theta = theta + (theta < 0) * 180;

        ns[tidx][tidy] = grad;

        __syncthreads();

        // Part 2: find edges

        if (tidx >= 2 && tidy >= 2 && tidx <= blockDim.x-3 && tidy <= blockDim.y-3) {    
            float max = 1;

            if (grad < lowThresh) {
                max = 0;
            }
    
            if ( (theta < 22) || (theta > 157) ) {
                if (ns[tidx-1][tidy] > grad)
                    max = 0;
                if (ns[tidx+1][tidy] > grad)
                    max = 0;
            }
            else if ( theta < 67 ) {
                if (ns[tidx-1][tidy-1] > grad)
                    max = 0;
                if (ns[tidx+1][tidy+1] > grad)
                    max = 0;
            }
            else if ( theta < 112 ) {
                if (ns[tidx][tidy-1] > grad)
                    max = 0;
                if (ns[tidx][tidy+1] > grad)
                    max = 0;
            }
            else {
                if (ns[tidx+1][tidy-1] > grad)
                    max = 0;
                if (ns[tidx-1][tidy+1] > grad)
                    max = 0;
            }
    
            if (col < width && row < height)
                edgemap[index] = max;
            
        }
    }
}

__device__ float fastatan2f(float a, float b) {
    if (fabs(b - 0.0001f) > 0)
        b = 0.001f;
    double x = a / b;
    x = x * x;
    return (float)((0.077650 * x - 0.287434) * x + 0.9951816) * x;
}

__device__ float fastsinf(float a) {
    return a;
}

__device__ float fastcosf(float a) {
    return -0.4 * a * a + 1;
}

__global__ void cannyEdge_v4(float *imageIn, float *edgemap, int width, int height, int lowThresh) {
    // Part 1: calculate gradient and angle

    __shared__ uint16_t ns[16][32];
    __shared__ uint16_t grads[16][32];

    int col = (threadIdx.x + blockIdx.x * blockDim.x) - (4 * blockIdx.x) - 2;
    int row = (threadIdx.y + blockIdx.y * blockDim.y) - (4 * blockIdx.y) - 2;
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int index = row * width + col;

    if (col >= 0 && row >= 0 && col < width && row < height)
        ns[tidx][tidy] = (uint16_t)(imageIn[index] * 100);
    else
        ns[tidx][tidy] = 0;

    __syncthreads();

    if (tidx >= 1 && tidy >= 1 && tidx <= blockDim.x-2 && tidy <= blockDim.y-2) {

        int gx = (int)ns[tidx+1][tidy] - (int)ns[tidx-1][tidy];
        int gy = (int)ns[tidx][tidy+1] - (int)ns[tidx][tidy-1];

        uint16_t grad = ( gx * gx + gy * gy );
        float theta = fastatan2f( gy, gx );

        grads[tidx][tidy] = grad;

        __syncthreads();

        // Part 2: find edges

        if (tidx >= 2 && tidy >= 2 && tidx <= blockDim.x-3 && tidy <= blockDim.y-3 && col < width && row < height && grad >= lowThresh) {    
            int x = (int)(fastcosf(theta) - 0.6) + (int)(fastcosf(theta) + 0.6);
            int y = (int)(fastsinf(theta) - 0.6) + (int)(fastsinf(theta) + 0.6);

            int max = fmaxf(grads[tidx + x][tidy + y], grad);
            max = fmaxf( grads[tidx - x][tidy - y], max );
            edgemap[index] = (max == grad);
        }
    }
}

__global__ void cannyEdge_v3_byte(uint8_t *imageIn, uint8_t *edgemap, int width, int height, int lowThresh) {

    // // 0 1 2
    // // 3   4
    // // 5 6 7

    // Part 1: calculate gradient and angle

    __shared__ int ns[32][8];

    int col = (threadIdx.x + blockIdx.x * blockDim.x) - (4 * blockIdx.x) - 2;
    int row = (threadIdx.y + blockIdx.y * blockDim.y) - (4 * blockIdx.y) - 2;
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int index = row * width + col;

    if (col >= 0 && row >= 0 && col < width && row < height)
        ns[tidx][tidy] = (int)(imageIn[index]);
    else
        ns[tidx][tidy] = 0;

    __syncthreads();

    if (tidx >= 1 && tidy >= 1 && tidx <= blockDim.x-2 && tidy <= blockDim.y-2) {

        int gx = -ns[tidx-1][tidy-1] + ns[tidx+1][tidy-1] - (2 * ns[tidx-1][tidy]) + (2 * ns[tidx+1][tidy]) - ns[tidx-1][tidy+1] + ns[tidx+1][tidy+1];
        int gy = -ns[tidx-1][tidy-1] + ns[tidx-1][tidy+1] - (2 * ns[tidx][tidy-1]) + (2 * ns[tidx][tidy+1]) - ns[tidx+1][tidy-1] + ns[tidx+1][tidy+1];

        int grad = (int)(sqrtf( powf(gx, 2) + powf(gy, 2) ));
        int theta = (int)(atan2f( gy, gx ) * 180 / M_PI);
        theta = theta + (theta < 0) * 180;

        __syncthreads();

        ns[tidx][tidy] = grad;

        __syncthreads();

        // Part 2: find edges
    
        if (tidx >= 2 && tidy >= 2 && tidx <= blockDim.x-3 && tidy <= blockDim.y-3) {    
            uint8_t max = 1;
    
            if ( (theta < 22) || (theta > 157) ) {
                if (ns[tidx-1][tidy] > grad)
                    max = 0;
                if (ns[tidx+1][tidy] > grad)
                    max = 0;
            }
            else if ( theta < 67 ) {
                if (ns[tidx-1][tidy-1] > grad)
                    max = 0;
                if (ns[tidx+1][tidy+1] > grad)
                    max = 0;
            }
            else if ( theta < 112 ) {
                if (ns[tidx][tidy-1] > grad)
                    max = 0;
                if (ns[tidx][tidy+1] > grad)
                    max = 0;
            }
            else {
                if (ns[tidx+1][tidy-1] > grad)
                    max = 0;
                if (ns[tidx-1][tidy+1] > grad)
                    max = 0;
            }
    
            if (grad < lowThresh) {
                max = 0;
            }
    
            if (col < width && row < height)
                edgemap[index] = max;
            
        }
    }
}

///////////////////////// applyMask
__global__ void applyMask_v0(float* inEdgemap, float* outMasked, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= width || idy >= height) return;

    // Define the points of the triangle (bottom left corner, bottom right corner, center of image)
    float x1 = 0;
    float y1 = height - 1;

    float x2 = width - 1;
    float y2 = height - 1;

    float x3 = (float)width / 2;
    float y3 = (float)height / 2;

    // Compute the barycentric coordinates of the current pixel
    float alpha = ((y2 - y3)*(idx - x3) + (x3 - x2)*(idy - y3)) / ((y2 - y3)*(x1 - x3) + (x3 - x2)*(y1 - y3));
    float beta = ((y3 - y1)*(idx - x3) + (x1 - x3)*(idy - y3)) / ((y2 - y3)*(x1 - x3) + (x3 - x2)*(y1 - y3));
    float gamma = 1.0 - alpha - beta;

    // Check if the current pixel is inside the triangle
    if (alpha >= 0.0 && beta >= 0.0 && gamma >= 0.0 && inEdgemap[idy * width + idx] > 0.5f) {
        outMasked[idy * width + idx] = 1.0;
    } else {
        outMasked[idy * width + idx] = 0.0;
    }
}

__global__ void applyMask_v1(float* inEdgemap, float* outMasked, int width, int height,
                               float wX, float hY, float denom) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= width || idy >= height) return;

    // Compute variables & global index
    float xA = idx - (wX + 1);
    float yA = idy - (hY + 1);
    int index = idy * width + idx;

    // Compute the barycentric coordinates of the current pixel
    float alpha = (hY * xA - wX * yA) / denom;
    float beta =  (-hY * xA - (wX+1) * yA) / denom;
    float gamma = 1.0 - alpha - beta;

    // Check if the current pixel is inside the triangle
    outMasked[index] = (alpha >= 0.0 && beta >= 0.0 && gamma >= 0.0 && inEdgemap[index] > 0.5f);
}

__global__ void applyMask_v1_byte(uint8_t* inEdgemap, float* outMasked, int width, int height,
    float wX, float hY, float denom) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int idy = blockIdx.y * blockDim.y + threadIdx.y;
if (idx >= width || idy >= height) return;

// Compute variables & global index
float xA = idx - (wX + 1);
float yA = idy - (hY + 1);
int index = idy * width + idx;

// Compute the barycentric coordinates of the current pixel
float alpha = (hY * xA - wX * yA) / denom;
float beta =  (-hY * xA - (wX+1) * yA) / denom;
float gamma = 1.0 - alpha - beta;

// Check if the current pixel is inside the triangle
outMasked[index] = (alpha >= 0.0 && beta >= 0.0 && gamma >= 0.0 && inEdgemap[index] == 1);
}

//Filter lines into positive and negative slope
__global__ void filterLines(float* lines, int numLines, int maxLines, float* posLines, float* negLines){
    bool negAdded = false;
    bool posAdded = true;

    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    //if thread is not in range of max lines then don't perform calculations
    if(tid >= numLines){
        return;
    }
        //get x1,y1,x2,y2,slope for line
        float x1 = lines[tid];
        float y1 = lines[tid + maxLines];
        float x2 = lines[tid + (maxLines * 2)];
        float y2 = lines[tid + (maxLines * 3)];
        float slope = lines[tid + (maxLines * 4)];

        float xdiff = x2 - x1;
        float ydiff = y2 - y1;

        //calculate length of line using sqrt((x2 - x1)^2 + (y2 - y1)^2)
        float lineLength = hypotf(xdiff, ydiff);

        //if line is long enough
        if(lineLength > 30){
            if(x1 != x2){

                //if positive sloped
                if(slope > 0){
                    float tanTheta = tanf(fabsf(ydiff)/fabsf(xdiff));
                    float angle = atanf(tanTheta) * (180/3.14);

                    //write line to posLines array
                    if(fabsf(angle) > 20 && fabsf(angle) < 85){
                        posLines[tid] = x1;
                        posLines[tid + maxLines] = y1;
                        posLines[tid + (maxLines * 2)] = x2;
                        posLines[tid + (maxLines * 3)] = y2;
                        posLines[tid + (maxLines * 4)] = slope;
                        posAdded = true;
                    }
                }

                //if negative sloped
                if(slope < 0){
                    float tanTheta = tanf(fabsf(ydiff)/fabsf(xdiff));
                    float angle = atanf(tanTheta) * (180/3.14);

                    //write line to negLines array
                    if(fabsf(angle) > 20 && fabsf(angle) < 85){
                        negLines[tid] = x1;
                        negLines[tid + maxLines] = y1;
                        negLines[tid + (maxLines * 2)] = x2;
                        negLines[tid + (maxLines * 3)] = y2;
                        negLines[tid + (maxLines * 4)] = slope;
                        negAdded = true;
                    }
                }
            }
        }

        //if no positive line added then perform above calculations again for line of any length
        if(!posAdded){
            if(slope > 0){
                    float tanTheta = tanf(fabsf(ydiff)/fabsf(xdiff));
                    float angle = atanf(tanTheta) * (180/3.14);

                    //write line to posLines array
                    if(fabsf(angle) > 20 && fabsf(angle) < 85){
                        posLines[tid] = x1;
                        posLines[tid + maxLines] = y1;
                        posLines[tid + (maxLines * 2)] = x2;
                        posLines[tid + (maxLines * 3)] = y2;
                        posLines[tid + (maxLines * 4)] = slope;
                        posAdded = true;
                    }
                }
            }

        if(!negAdded){
            if(slope < 0){
                    float tanTheta = tanf(fabsf(ydiff)/fabsf(xdiff));
                    float angle = atanf(tanTheta) * (180/3.14);

                    //write line to negLines array
                    if(fabsf(angle) > 20 && fabsf(angle) < 85){
                        negLines[tid] = x1;
                        negLines[tid + maxLines] = y1;
                        negLines[tid + (maxLines * 2)] = x2;
                        negLines[tid + (maxLines * 3)] = y2;
                        negLines[tid + (maxLines * 4)] = slope;
                        negAdded = true;
                    }
                }
            }
    }
